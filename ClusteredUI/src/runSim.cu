#include "hip/hip_runtime.h"
/*The MIT License (MIT)

Copyright (c) [2015] [Sawyer Hopkins]

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.*/

#include "ui.h"
#include <dlfcn.h>

using namespace std;
using namespace utilities;

__global__
void setupIntegrator(integrators::brownianIntegrator * difeq)
{
	difeq = new integrators::brownianIntegrator();
}

void initIntegrator(integrators::brownianIntegrator * difeq, configReader::config * cfg)
{
	//Allocate enough memory to hold this object in device memory.
	integrators::brownianIntegrator * dummy = new integrators::brownianIntegrator(cfg);
	hipMalloc((void **)&difeq, sizeof(dummy));
	//Setup the integrator.
	setupIntegrator<<<1,1>>>(difeq);
	hipDeviceSynchronize();
	
	string err = hipGetErrorString(hipGetLastError());
	if (err != "no error")
	{
		util::writeTerminal("CUDA KERNEL: setupIntegrator -" + err + "\n", Colour::Red);
	}
	else
	{
		util::writeTerminal("CUDA KERNEL: setupIntegrator -" + err + "\n", Colour::Green);
	}
	
	//printf("CUDA KERNEL: setupIntegrator -\t%s\n\n", hipGetErrorString(hipGetLastError()));
}

void initForces(void* forceLib, physics::IForce * force, configReader::config * cfg)
{
	//Throw error if the library does not exist.
	if (!forceLib)
	{
		util::writeTerminal("\n\nError loading in force library.\n\n", Colour::Red);
		return;
	}

	dlerror();

	//Make a factory to create the force instance.
	physics::create_Force* factory = (physics::create_Force*) dlsym(forceLib,"getForce");
	const char* err = dlerror();

	//If the force is not properly implemented.
	if (err)
	{
		util::writeTerminal("\n\nCould not find symbol: getForce\n\n", Colour::Red);
		return;
	}

	//Create a new force instance from the factory.
	physics::IForce* dummy = factory(cfg);
	//Allocate enough memory to hold this object in device memory.
	hipMalloc((void **)&force, sizeof(dummy));

	//Make a factory to create the force instance.
	physics::create_cudaForce* cudaFactory = (physics::create_cudaForce*) dlsym(forceLib,"getCudaForce");
	err = dlerror();

	//If the force is not properly implemented.
	if (err)
	{
		util::writeTerminal("\nCould not find symbol: getCudaForce", Colour::Red);
		return;
	}

	cudaFactory(force);

	physics::cuda_test* testFactory = (physics::cuda_test*) dlsym(forceLib,"getCudaTest");

	testFactory(force);
}

/**
 * @brief Run a new simulation.
 */
void runScript()
{
	/*----------------CFG-----------------*/

	util::writeTerminal("Looking for configuration file.\n\n", Colour::Green);
	configReader::config * cfg =new configReader::config("settings.cfg");
	cfg->showOutput();

	/*-------------INTEGRATOR-------------*/

	//Create the integrator.
	util::writeTerminal("Creating integrator.\n", Colour::Green);
	integrators::brownianIntegrator * difeq;
	initIntegrator(difeq, cfg);

	/*---------------FORCES---------------*/

	//Create the force.
	util::writeTerminal("Creating forces.\n", Colour::Green);
	std::string forceName = cfg->getParam<std::string>("force","");
	std::string fileName = "./" + forceName + ".so";
	physics::IForce * loadForce;
	void* forceLib = dlopen(fileName.c_str(), RTLD_LAZY);
	//initForces(forceLib, loadForce, cfg);

	/*---------------SYSTEM---------------*/

	util::writeTerminal("\nCreating particle system.\n", Colour::Green);

	//Set the number of particles.
	int nParticles = cfg->getParam<int>("nParticles",0);

	if (nParticles == 0)
	{
		util::writeTerminal("\n\nSystem must start with more than zero particles.", Colour::Red);
		return;
	}
	//Creates the particle system.
	simulation::system * sys = new simulation::system(cfg, difeq, loadForce, forceLib, nParticles);

	/*---------------RUNNING--------------*/

	//Output the stats.
	cout << "---Number of Particles: " << sys->getNParticles() << "\n";
	cout << "---Box Size: " << sys->getBoxSize() << "\n";
	cout << "---Cell Size: " << sys->getCellSize() << "\n\n";
 
	//Write the initial system.
	cout << "Writing initial system to file.\n\n";
	sys->writeSystem("/initSys");

	/*-------------Iterator-------------*/

	//Allow user to check system settings before running.
	//Comment this section out if running without terminal access.
	util::writeTerminal("System initialization complete. Press y/n to continue: ", Colour::Blue);
	std::string cont;
	cin >> cont;

	if (cont != "Y" && cont != "y")
	{
		exit(100);
	}

	util::writeTerminal("Starting integration.\n", Colour::Green);

	int endTime = cfg->getParam<float>("endTime",1000);

	sys->run(endTime);

	//Write the final system.
	util::writeTerminal("\nIntegration complete.\n\n Writing final system to file.", Colour::Green);
	sys->writeSystem("/finSys");
}