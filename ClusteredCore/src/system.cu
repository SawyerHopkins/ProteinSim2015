#include "hip/hip_runtime.h"
/*The MIT License (MIT)

Copyright (c) [2015] [Sawyer Hopkins]

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.*/

#include "system.h"
#include "globals.cuh"

namespace simulation
{
	system::system(configReader::config* cfg, integrators::I_integrator* sysInt, physics::IForce* sysFcs, int nParts) : particles(new particle[nParts])
	{
		/********************************************//**
		*------------------SETUP OUTPUT------------------
		************************************************/

		//Sets the trial name
		trialName = cfg->getParam<std::string>("trialName", "");

		if (trialName == "")
		{
			runSetup();
		}
		else
		{
			//Check that the provided directory exists.
			bool validDir = checkDir(trialName);
			if (validDir == true)
			{
				utilities::util::writeTerminal("\nTrial name already exists. Overwrite (y,n): ", utilities::Colour::Magenta);

				//Check user input
				std::string cont;
				std::cin >> cont;

				if (cont != "Y" && cont != "y")
				{
					runSetup();
				}
			}
			else
			{
				//Attempt to make the directory.
				mkdir(trialName.c_str(),0777);

				//Check that we were able to make the desired directory.
				validDir = checkDir(trialName);
				if (validDir == false)
				{
					runSetup();
				}
			}

		}

		/********************************************//**
		*-------------------LOAD INPUT-------------------
		************************************************/

		//Set time information
		currentTime = 0;
		dTime = cfg->getParam<double>("timeStep",0.001);

		//Set the random number generator seed.
		seed = cfg->getParam<int>("seed",90210);

		//Sets the system temperature.
		temp = cfg->getParam<double>("temp",1.0);

		//Set the number of particles.
		nParticles = cfg->getParam<int>("nParticles",1000);

		//How often to output snapshots.
		outputFreq = cfg->getParam<int>("outputFreq",int(1.0/dTime));

		//Option to output XYZ format for clusters
		outXYZ = cfg->getParam<int>("XYZ",0);

		//Set the integration method.
		integrator = sysInt;

		//Set the internal forces.
		sysForces = sysFcs;

		//Set the concentration.
		double conc = cfg->getParam<double>("conc",0.01);

		//Set the radius.
		double r = cfg->getParam<double>("radius",0.5);

		//Set the mass.
		double m = cfg->getParam<double>("mass",1.0);

		//Set the scale.
		int scale = 0;
		scale = cfg->getParam<int>("scale",4);

		//Create a box based on desired concentration.
		double vP = nParticles*(4.0/3.0)*atan(1.0)*4.0*r*r*r;
		boxSize = (int) cbrt(vP / conc);

		//Calculates the number of cells needed.
		cellSize = boxSize / scale;
		boxSize = cellSize * scale;
		cellScale = scale;
		numCells = pow(scale,3.0);

		//Sets the actual concentration.
		concentration = vP/pow(boxSize,3.0);
		particlesPerCell = 10;

		std::cout << "---System concentration: " << concentration << "\n";

		/********************************************//**
		*-----------------COPY VARIABLES-----------------
		************************************************/

		//Copy over system variables to device.
		hipMalloc((void **)&d_nParticles, sizeof(int));
		hipMemcpy(d_nParticles, &nParticles, sizeof(int), hipMemcpyHostToDevice);

		hipMalloc((void **)&d_boxSize, sizeof(int));
		hipMemcpy(d_boxSize, &boxSize, sizeof(int), hipMemcpyHostToDevice);

		hipMalloc((void **)&d_dTime, sizeof(double));
		hipMemcpy(d_dTime, &dTime, sizeof(double), hipMemcpyHostToDevice);

		hipMalloc((void **)&d_currentTime, sizeof(double));
		hipMemcpy(d_currentTime, &currentTime, sizeof(double), hipMemcpyHostToDevice);

		hipMalloc((void **)&d_cellScale, sizeof(int));
		hipMemcpy(d_cellScale, &cellScale, sizeof(int), hipMemcpyHostToDevice);

		hipMalloc((void **)&d_cellSize, sizeof(int));
		hipMemcpy(d_cellSize, &cellSize, sizeof(int), hipMemcpyHostToDevice);

		hipMalloc((void **)&d_numCells, sizeof(int));
		hipMemcpy(d_numCells, &numCells, sizeof(int), hipMemcpyHostToDevice);

		hipMalloc((void **)&d_particlesPerCell, sizeof(int));
		hipMemcpy(d_particlesPerCell, &particlesPerCell, sizeof(int), hipMemcpyHostToDevice);

		hipMalloc((void *****)&cells, sizeof(cell****));
		/********************************************//**
		*----------------CREATE PARTICLES----------------
		************************************************/

		initParticles(r,m);

		/********************************************//**
		*------------------CREATE CELLS------------------
		************************************************/

		dim3 cellBlocks(scale,scale,scale);

		//Create cells.
		initCells<<<1,1>>>(numCells, cellScale, cells, particlesPerCell);
		hipDeviceSynchronize();
		printf("Device Variable Copying:\t%s\n", hipGetErrorString(hipGetLastError()));

		resetCells<<<cellBlocks,1>>>(cells);
		hipDeviceSynchronize();
		printf("Device Variable Copying:\t%s\n", hipGetErrorString(hipGetLastError()));

		updateCells<<<numCells,1>>>(d_cellScale, d_cellSize, cells, d_particles);
		hipDeviceSynchronize();
		printf("Device Variable Copying:\t%s\n", hipGetErrorString(hipGetLastError()));

		std::cout << "Created: " << numCells << " cells from scale: " <<  cellScale << "\n";
		writeSystemInit();

	}

	void system::run(double endTime)
	{
		cycleHour = (endTime / dTime) / 3600.0;
		//Create the snapshot name.
		std::string snap = trialName + "/snapshots";
		mkdir(snap.c_str(),0777);

		//Create the movie folder
		std::string mov = trialName + "/movie";
		mkdir(mov.c_str(),0777);

		//Debugging counter.
		int counter = 0;

		//Diagnostics timer.
		debugging::timer* tmr = new debugging::timer();
		tmr->start();

		//Run system until end time.
		while (currentTime < endTime)
		{
			//Get the forces acting on the system.
			getAcceleration<<<nParticles,1>>>(d_nParticles,d_boxSize,d_currentTime,cells,d_particles,d_sysForces);
			printf("Device Variable Copying:\t%s\n", hipGetErrorString(hipGetLastError()));
			//hipDeviceSynchronize();
			//Get the next system.
			//nextSystem<<<nParticles,1>>>(d_currentTime, d_dTime, d_nParticles, d_boxSize, cells, d_particles, d_sysForces, d_integrator);
			//hipDeviceSynchronize();
			//Call cell manager.
			///resetCells<<<cellScale,1>>>(cells);
			//hipDeviceSynchronize();
			//updateCells<<<numCells,1>>>(d_cellScale, d_cellSize, cells, d_particles);
			hipDeviceSynchronize();
			printf("Device Variable Copying:\t%s\n", hipGetErrorString(hipGetLastError()));
			//Output a snapshot every second.
			if ( (counter % outputFreq) == 0 )
			{
				hipMemcpy(particles,d_particles, nParticles*sizeof(particle) ,hipMemcpyDeviceToHost);
				printf("Device Variable Copying:\t%s\n", hipGetErrorString(hipGetLastError()));
				int q = 0;
				std::cin >> q;
				if (currentTime > 0)
				{
					utilities::util::clearLines(13);
				}
				writeSystemState(tmr);
			}

			//Update loading bar.
			utilities::util::loadBar(currentTime,endTime,counter);

			//Increment counters.
			currentTime += dTime;
			counter++;
		}
	}
}