#include "hip/hip_runtime.h"
/*The MIT License (MIT)

Copyright (c) [2015] [Sawyer Hopkins]

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.*/

#include "integrator.h"

namespace integrators
{

	__global__
	void brownianIntegrator::nextSystem(double *time, double *dt, int *nParticles, int *boxSize, simulation::cell**** cells, simulation::particle** items, physics::forces* f)
	{
		//Checks what method is needed.
		if (time == 0)
		{
			firstStep(time, dt, nParticles, boxSize, items, f);
		}
		else
		{
			normalStep(time, dt, nParticles, boxSize, items, f);
		}
	}

	__device__
	void brownianIntegrator::firstStep(double time, double dt, int nParticles, int boxSize, simulation::particle** items, physics::forces* f)
	{
		//Add 4 threads to the team.
		for (int i=0; i < nParticles; i++)
		{

			//SEE GUNSTEREN AND BERENDSEN 1981 EQ 2.26

			memCorrX[i] = 0.0;
			memCorrY[i] = 0.0;
			memCorrZ[i] = 0.0;

			memX[i] = (*Dist)(*gen);
			memY[i] = (*Dist)(*gen);
			memZ[i] = (*Dist)(*gen);

			double m = 1.0/items[i]->getMass();
			double xNew = items[i]->getX() + (items[i]->getVX() * coEff1 * dt) + (items[i]->getFX() * coEff3 * dt * dt * m) + (sig1 * memX[i]);
			double yNew = items[i]->getY() + (items[i]->getVY() * coEff1 * dt) + (items[i]->getFY() * coEff3 * dt * dt * m) + (sig1 * memY[i]);
			double zNew = items[i]->getZ() + (items[i]->getVZ() * coEff1 * dt) + (items[i]->getFZ() * coEff3 * dt * dt * m) + (sig1 * memZ[i]);
			items[i]->setPos(xNew,yNew,zNew,boxSize);

		}
	}

	__device__
	void brownianIntegrator::normalStep(double time, double dt, int nParticles, int boxSize, simulation::particle** items, physics::forces* f)
	{

		double dt2 = dt * dt;
		//SEE GUNSTEREN AND BERENDSEN 1981 EQ 2.26
		//New random walk.
		memCorrX[i] = (*Dist)(*tgens[omp_get_thread_num()]);
		memCorrY[i] = (*Dist)(*tgens[omp_get_thread_num()]);
		memCorrZ[i] = (*Dist)(*tgens[omp_get_thread_num()]);

		//Correlation to last random walk.
		memCorrX[i] = sig2 * ((corr * memX[i])+(dev * memCorrX[i]));
		memCorrY[i] = sig2 * ((corr * memY[i])+(dev * memCorrY[i]));
		memCorrZ[i] = sig2 * ((corr * memZ[i])+(dev * memCorrZ[i]));

		memX[i] = (*Dist)(*tgens[omp_get_thread_num()]);
		memY[i] = (*Dist)(*tgens[omp_get_thread_num()]);
		memZ[i] = (*Dist)(*tgens[omp_get_thread_num()]);

		double m = 1.0/items[i]->getMass();

		double x0 = items[i]->getX0();
		double y0 = items[i]->getY0();
		double z0 = items[i]->getZ0();

		//Run the integration routine.
		double xNew = ((1.0+coEff0) * items[i]->getX());
		xNew -= (coEff0 * x0);
		xNew += (m * dt2 * coEff1 * items[i]->getFX());
		xNew += (m * dt2 * coEff2 * (items[i]->getFX() - items[i]->getFX0()));
		xNew += (sig1 * memX[i]) + (coEff0 * memCorrX[i]);

		double yNew = ((1.0+coEff0) * items[i]->getY()) ;
		yNew -= (coEff0 * y0);
		yNew += (m * dt2 * coEff1 * items[i]->getFY());
		yNew += (m * dt2 * coEff2 * (items[i]->getFY() - items[i]->getFY0()));
		yNew += (sig1 * memY[i]) + (coEff0 * memCorrY[i]);

		double zNew = ((1.0+coEff0) * items[i]->getZ());
		zNew -= (coEff0 * z0);
		zNew += (m * dt2 * coEff1 * items[i]->getFZ());
		zNew += (m * dt2 * coEff2 * (items[i]->getFZ() - items[i]->getFZ0()));
		zNew += (sig1 * memZ[i]) + (coEff0 * memCorrZ[i]);

		//Velocity is not needed for brownianIntegration.
		//Run velocity integration at the same frequency as
		//the temperature/energy analysis routine.
		//-------------------------------------------------
		//For best perfomance use
		//velFreq = outputFreq.
		//-------------------------------------------------
		//If using a velocity dependant force use
		//velFreq = 0.
		//-------------------------------------------------
		//For all other cases do whatever.
		if (velFreq == 0)
		{
			velocityStep(items, i, xNew, yNew, zNew, dt, boxSize);
		}
		else if (velCounter == velFreq)
		{
			velocityStep(items, i, xNew, yNew, zNew, dt, boxSize);
		}

		items[i]->setPos(xNew, yNew, zNew, boxSize);

		//Manage velocity output counter.
		if (velCounter == velFreq)
		{
			velCounter = 0;
		}
		else
		{
			velCounter++;
		}
	}

	__device__
	void brownianIntegrator::velocityStep(simulation::particle** items, int i, double xNew0, double yNew0, double zNew0, double dt, double boxSize)
	{

		double m = 1.0/items[i]->getMass();

		//Current position and previous position are already PBC safe.
		//Their difference is also already PBC safe.
		double dx0 = items[i]->getX() - items[i]->getX0();
		double dy0 = items[i]->getY() - items[i]->getY0();
		double dz0 = items[i]->getZ() - items[i]->getZ0();

		//Make the new position PBC safe.
		double xNew = utilities::util::safeMod(xNew0,boxSize);
		double yNew = utilities::util::safeMod(yNew0,boxSize);
		double zNew = utilities::util::safeMod(zNew0,boxSize);

		//Make the difference between the new position and the current position PBC safe.
		double x0 = utilities::util::safeMod0(items[i]->getX(), xNew, boxSize);
		double y0 = utilities::util::safeMod0(items[i]->getY(), yNew, boxSize);
		double z0 = utilities::util::safeMod0(items[i]->getZ(), zNew, boxSize);

		//Take the difference.
		double dx = xNew - x0;
		double dy = yNew - y0;
		double dz = zNew - z0;

		//Precompute.
		double dt2 = dt * dt;
		double dt3 = dt * dt2;

		//Run the integration routine.
		double vxNew = dx + dx0;
		vxNew += (m * dt2 * goy2 * items[i]->getFX());
		vxNew -= (m * dt3 * goy3 * (items[i]->getFX() - items[i]->getFX0()));
		vxNew += (memCorrX[i] - sig1*memX[i]);
		vxNew *= (hn * dtInv);

		double vyNew = dy + dy0;
		vyNew += (m * dt2 * goy2 * items[i]->getFY());
		vyNew -= (m * dt3 * goy3 * (items[i]->getFY() - items[i]->getFY0()));
		vyNew += (memCorrY[i] - sig1*memY[i]);
		vyNew *= (hn * dtInv);

		double vzNew = dz + dz0;
		vzNew += (m * dt2 * goy2 * items[i]->getFZ());
		vzNew -= (m * dt3 * goy3 * (items[i]->getFZ() - items[i]->getFZ0()));
		vzNew += (memCorrZ[i] - sig1*memZ[i]);
		vzNew *= (hn * dtInv);

		//Set the velocities.
		items[i]->setVX(vxNew);
		items[i]->setVY(vyNew);
		items[i]->setVZ(vzNew);

	}
}