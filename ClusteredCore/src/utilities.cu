#include "hip/hip_runtime.h"
/*The MIT License (MIT)

Copyright (c) [2015] [Sawyer Hopkins]

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.*/

#include "utilities.h"

namespace utilities
{
	__host__ __device__
	double util::safeMod(double val, double base)
	{
		//0 mod n is always zero
		if (val == 0)
		{
			return 0.0;
		}
		//if the particle is on the edge of the system.
		else if (val == base)
		{
			return 0.0;
		}
		//if the particle is outside the upper bounds.
		else if (val>base)
		{
			return (val-base);
		}
		//if the particle is outside the lower bounds.
		else if (val<0)
		{
			return (val+base);
		}
		//No problems return value.
		else
		{
			return val;
		}
	}

	__host__ __device__
	double util::safeMod0(double val0, double val, double base)
	{
		//The difference between the two values.
		double dx = val - val0;
		//If the values are further apart than half the system, use PBC.
		if (fabs(dx) > base/2 )
		{
			//Check which direction to implement PBC.
			if (dx < 0)
			{
				return val0-base;
			}
			else
			{
				return val0+base;
			}
		}
		else
		{
			return val0;
		}
	}

	__device__
	double util::pbcDist(double X,double Y, double Z,double X1, double Y1,double Z1,double L)
	{

		double dx = fabs(X-X1);
		double dy = fabs(Y-Y1);
		double dz = fabs(Z-Z1);

		//Check X direction.
		if(dx > L/2 )
		{
			dx-=L;
		}

		//Check Y direction.
		if(dy > L/2 )
		{
			dy-=L;
		}

		//Check Z direction.
		if(dz > L/2 )
		{
			dz-=L;
		}
		//Pythag for the distance.
		return (dx*dx)+(dy*dy)+(dz*dz);

	}

	__device__
	void util::unitVectorSimple(double dX, double dY, double dZ, double r, double (&acc)[3])
	{
		//Normalize by distance.
		acc[0]=dX/r;
		acc[1]=dY/r;
		acc[2]=dZ/r;
	}

	__device__
	void util::unitVectorAdv(double X,double Y, double Z,double X1, double Y1,double Z1,double (&acc)[3],double r,int L)
	{
		double dx,dy,dz;

		dx=X1-X; dy=Y1-Y; dz=Z1-Z;

		//Check X PBC.
		if(fabs(dx) > L/2)
		{
			if(dx<0)
			{
				dx=dx+L;
			}
			else
			{
				dx=dx-L;
			}
		}

		//Check Y PBC.
		if(fabs(dy) > L/2)
		{
			if(dy<0)
			{
				dy=dy+L;
			}
			else
			{
				dy=dy-L;
			}
		}

		//Check Z PBC.
		if(fabs(dz) > L/2)
		{
			if(dz<0)
			{
				dz=dz+L;
			}
			else
			{
				dz=dz-L;
			}
		}

		//Normalize by distance.
		dx=dx/r; dy=dy/r; dz=dz/r;
		acc[0]=dx; acc[1]=dy; acc[2]=dz;
	}

	__device__
	double util::powBinaryDecomp(double base, int exp)
	{
		double answer = 1;
		while(exp)
		{
			if (exp & 1)
			{
				answer *= base;
			}
			exp >>= 1;
			base *= base;
		}
		return answer;
	}
}