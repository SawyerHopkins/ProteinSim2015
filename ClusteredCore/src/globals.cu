#include "hip/hip_runtime.h"
/*The MIT License (MIT)

Copyright (c) <2015> <Sawyer Hopkins>

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in
all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
THE SOFTWARE.*/

#include "force.h"

namespace physics
{
	/********************************************//**
	*-----------------FORCE MANAGEMENT---------------
	 ***********************************************/
	/**
	 * @brief Find the net force on all particles in the system.  
	 * @param nPart The number of particles in the system.
	 * @param boxSize The size of the system.
	 * @param time The current system time.
	 * @param cells The system cell manager.
	 * @param items The particles in the system.
	 */
	__global__
	void getAcceleration(int *nPart, int *boxSize, double *time, simulation::cell**** cells, simulation::particle** items, IForce* flist)
	{
		int index = 0;
		for (index; index < nPart; index++)
		{
			//Resets the force on the particle.
			items[index]->nextIter();

			simulation::particle* p = items[index];
			simulation::cell* itemCell = cells[p->getCX()][p->getCY()][p->getCZ()];

			//Iterates through all forces.
			flist->getAcceleration(index, *nPart, *boxSize, *time, itemCell, items);
		}
	}

}

namespace integrators
{

	__global__
	void brownianIntegrator::nextSystem(double *time, double *dt, int *nParticles, int *boxSize, simulation::cell**** cells, simulation::particle** items, physics::forces* f, I_integrator* inter)
	{
		inter->nextSystem(time, dt, nParticles, boxSize, cells, items, f)
	}

}

namespace simulation
{
	/**
	 * @brief Creates the cell system.
	 * @param numCells The number of cells to be created.
	 * @param scale The number of cells in each dimension. (numCells^1/3)
	 */
	__global__
	void system::initCells(int numCells, int scale, simulation::cell**** cells, simulation::particle** d_particles)
	{

		//Create the cells.
		cells = new cell***[scale];
		for(int i=0; i < scale; i++)
		{
			cells[i] = new cell**[scale];
			for(int j=0; j < scale; j++)
			{
				cells[i][j] = new cell*[scale];
				for(int k=0; k < scale; k++)
				{
					cells[i][j][k] = new cell();
				}
			}
		}

		//Set the cell neighbors.
		for(int x=0; x < scale; x++)
		{
			for(int y=0; y < scale; y++)
			{
				for(int z=0; z < scale; z++)
				{
					int left = x-1;
					int right = x+1;
					int top = y-1;
					int bot = y+1;
					int front = z-1;
					int back = z+1;

					if (x == 0)
					{
						left = (scale-1);
					}
					else if (x == (scale-1))
					{
						right = 0;
					}

					if (y == 0)
					{
						top = (scale-1);
					}
					else if (y == (scale-1))
					{
						bot = 0;
					}

					if (z == 0)
					{
						front = (scale-1);
					}
					else if (z == (scale-1))
					{
						back = 0;
					}

					//Set the 6 principle cells next the current cell.
					//Access diagonals through combinations of these six.
					cells[x][y][z]->left = cells[left][y][z];
					cells[x][y][z]->right = cells[right][y][z];
					cells[x][y][z]->top = cells[x][top][z];
					cells[x][y][z]->bot = cells[x][bot][z];
					cells[x][y][z]->front = cells[x][y][front];
					cells[x][y][z]->back = cells[x][y][back];
				}
			}
		}

		//Check the cell neighborhood mapping.
		for(int x=0; x < scale; x++)
		{
			for(int y=0; y < scale; y++)
			{
				for(int z=0; z < scale; z++)
				{
					cells[x][y][z]->createNeighborhood();
				}
			}
		}

		//Assign the particle to their starting cell.
		for(int i=0; i < nParticles; i++)
		{
			int cx = d_particles[i]->getX() / cellSize;
			int cy = d_particles[i]->getY() / cellSize;
			int cz = d_particles[i]->getZ() / cellSize;

			//Tell the particle what cell its in, then add to cell.
			d_particles[i]->setCell(cx,cy,cz);
			cells[cx][cy][cz]->addMember(&(d_particles[i]));

		}

	}
}